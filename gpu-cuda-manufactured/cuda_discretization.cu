#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "cuda_data.h"
#include "boundaries.h"
#include "numerics.h"
#include "mesh.h"
#include "timer.h"
}

#include "cuda_kernels.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_lap,
                                   const int nx, const int ny, const int nm)
{
	int dst_x, dst_y, dst_nx, dst_ny;
	int src_x, src_y, src_nx, src_ny;
	int til_x, til_y, til_nx;
	fp_t value=0.;

	/* source and tile width include the halo cells */
	src_nx = blockDim.x;
	src_ny = blockDim.y;
	til_nx = src_nx;

	/* destination width excludes the halo cells */
	dst_nx = src_nx - nm + 1;
	dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	til_x = threadIdx.x;
	til_y = threadIdx.y;

	dst_x = blockIdx.x * dst_nx + til_x;
	dst_y = blockIdx.y * dst_ny + til_y;

	src_x = dst_x - nm/2;
	src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		if (dst_y < ny && dst_x < nx) {
			d_conc_lap[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__device__ void source_stvdwtt(const fp_t x,  const fp_t y,  const fp_t y,
                               const fp_t A1, const fp_t A2, const fp_t B1, const fp_t B2,
                               const fp_t C2, const fp_t kappa,
                               fp_t* S)
{
    fp_t alpha = 0.;
    manufactured_shift(x, t, A1, A2, B1, B2, C2, &alpha);
    const fp_t dadx = A1 * B1 * t * cos(B1 * x) + A2 * B2 * cos(B2 * x + C2 * t);
    const fp_t d2adx2 = -A1 * B1 * B1 * t * sin(B1 * x) - A2 * B2 * B2 * sin(B2 * x + C2 * t);
    const fp_t dadt = A1 * sin(B1 * x) + A2 * C2 * cos(B2 * x + C2 * t);
    const fp_t Q = (y - alpha) / sqrt(2. * kappa);
    const fp_t sech = 1.0 / cosh(Q);
    const fp_t sum = -sqrt(4. * kappa) * tanh(Q) * dadx * dadx + sqrt(2.) * (dadt - kappa * d2adx2);
    *S = sech * sech / sqrt(16. * kappa) * sum;
}

__device__ void source_sympy(const fp_t x,  const fp_t y,  const fp_t y,
                               const fp_t A1, const fp_t A2, const fp_t B1, const fp_t B2,
                               const fp_t C2, const fp_t kappa,
                               fp_t* S)
{
    const fp_t Q = tanh((1.0L/2.0L)*sqrt(2)*(A1*t*sin(B1*x) + A2*sin(B2*x + C2*t) - y + 0.25)/sqrt(kappa);
    S_result = (  0.5  * sqrt(kappa)*tanh((1.0L/2.0L)*sqrt(2)*(A1*t*sin(B1*x) + A2*sin(B2*x + C2*t) - y + 0.25)/sqrt(kappa))
                - 0.25 * sqrt(2)*(A1*sin(B1*x) + A2*C2*cos(B2*x + C2*t))
               ) * (Q * Q - 1)/sqrt(kappa);
}
        


__global__ void evolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new, fp_t* d_conc_lap,
                                 const int nx, const int ny, const int nm, const fp_t dt,
                                 const fp_t A1, const fp_t A2, const fp_t B1, const fp_t B2,
                                 const fp_t C2, const fp_t kappa)
{
	int thr_x, thr_y, x, y;

	/* determine indices on which to operate */
	thr_x = threadIdx.x;
	thr_y = threadIdx.y;

	x = blockDim.x * blockIdx.x + thr_x;
	y = blockDim.y * blockIdx.y + thr_y;

	/* explicit Euler solution to the Allen-Cahn equation of motion */
	if (x < nx && y < ny) {
		d_conc_new[nx * y + x] = d_conc_old[nx * y + x]
		              + dt * D * d_conc_lap[nx * y + x];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void device_boundaries(fp_t* conc,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    conc, nx, ny, nm
	);
}

void device_convolution(fp_t* conc_old, fp_t* conc_lap,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    conc_old, conc_lap, nx, ny, nm
	);

}

void device_evolution(fp_t* conc_old, fp_t* conc_new, fp_t* conc_lap,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by,
                        const fp_t D, const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	evolution_kernel<<<num_tiles,tile_size>>> (
	    conc_old, conc_new, conc_lap, nx, ny, nm, dt, A1, A2, B1, B2, C2, kappa
	);
}

void read_out_result(fp_t** conc, fp_t* d_conc, const int nx, const int ny)
{
	hipMemcpy(conc[0], d_conc, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}

void cuda_evolution_solver(struct CudaData* dev, fp_t** conc_new,
                           const int bx,  const int by,
                           const int nm,  const int nx, const int ny,
						   const fp_t A1, const fp_t A2,
						   const fp_t B1, const fp_t B2,
						   const fp_t C2, const fp_t kappa,
                           const fp_t dt, struct Stopwatch* sw)
{
	double start_time;

	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	/* apply boundary conditions */
	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_old, nx, ny, nm
	);

	/* compute Laplacian */
	start_time = GetTimer();
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_old, dev->conc_lap, nx, ny, nm
	);
	sw->conv += GetTimer() - start_time;

	/* compute result */
	start_time = GetTimer();
	diffusion_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_old, dev->conc_new, dev->conc_lap, nx, ny, nm, D, dt
	);
	sw->step += GetTimer() - start_time;
}
