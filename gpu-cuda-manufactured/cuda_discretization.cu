#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "cuda_data.h"
#include "boundaries.h"
#include "numerics.h"
#include "mesh.h"
#include "timer.h"
}

#include "cuda_kernels.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__device__ void device_manufactured_solution(const fp_t x,  const fp_t y,  const fp_t t,
                                             const fp_t A1, const fp_t A2,
                                             const fp_t B1, const fp_t B2,
                                             const fp_t C2, const fp_t kappa,
                                             fp_t* eta)
{
	/* Equation 3 */
	const fp_t alpha = 0.25 + A1 * t * sin(B1 * x) + A2 * sin(B2 * x + C2 * t);
    /* Equation 2 */
	*eta = 0.5 * (1. - tanh((y - alpha)/sqrt(2. * kappa)));
}

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_lap,
                                   const int nx, const int ny, const int nm)
{
	int dst_x, dst_y, dst_nx, dst_ny;
	int src_x, src_y, src_nx, src_ny;
	int til_x, til_y, til_nx;
	fp_t value=0.;

	/* source and tile width include the halo cells */
	src_nx = blockDim.x;
	src_ny = blockDim.y;
	til_nx = src_nx;

	/* destination width excludes the halo cells */
	dst_nx = src_nx - nm + 1;
	dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	til_x = threadIdx.x;
	til_y = threadIdx.y;

	dst_x = blockIdx.x * dst_nx + til_x;
	dst_y = blockIdx.y * dst_ny + til_y;

	src_x = dst_x - nm/2;
	src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		if (dst_y < ny && dst_x < nx) {
			d_conc_lap[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__device__ void source(const fp_t x,  const fp_t y, const fp_t t,
                       const fp_t A1, const fp_t A2,
                       const fp_t B1, const fp_t B2,
                       const fp_t C2, const fp_t kappa,
                       fp_t* S)
{
	/* Equation 3 */
	const fp_t alpha = 0.25 + A1 * t * sin(B1 * x) + A2 * sin(B2 * x + C2 * t);
    /* Equation 4 */
    const fp_t dadx = A1 * B1 * t * cos(B1 * x) + A2 * B2 * cos(B2 * x + C2 * t);
    const fp_t d2adx2 = -A1 * B1 * B1 * t * sin(B1 * x) - A2 * B2 * B2 * sin(B2 * x + C2 * t);
    const fp_t dadt = A1 * sin(B1 * x) + A2 * C2 * cos(B2 * x + C2 * t);
    const fp_t Q = (y - alpha) / sqrt(2. * kappa);
    const fp_t sech = 1. / cosh(Q);
    const fp_t sum = -sqrt(4. * kappa) * tanh(Q) * dadx * dadx + sqrt(2.) * (dadt - kappa * d2adx2);
    *S = sech * sech / sqrt(16. * kappa) * sum;
}

__device__ void source_sympy(const fp_t x,  const fp_t y,  const fp_t t,
                             const fp_t A1, const fp_t A2, const fp_t B1, const fp_t B2,
                             const fp_t C2, const fp_t kappa,
                             fp_t* S)
{
    const fp_t sq2 = sqrt(2.);
    const fp_t sqK = sqrt(kappa);
    const fp_t Q = 0.5*sq2*(-y + A1*t*sin(B1*x) + A2*sin(B2*x + C2*t) + 0.25)/sqK;
    const fp_t sech2 = 1. - tanh(Q)*tanh(Q);
    *S = (1. - tanh(Q)*tanh(Q))/sqrt(16.*kappa) * (2.0*sqK*pow(A1*B1*t*cos(B1*x) + A2*B2*cos(B2*x + C2*t), 2)*(-sech2)*tanh(Q)
                                                   + sq2*kappa*(A1*pow(B1, 2)*t*sin(B1*x) + A2*pow(B2, 2)*sin(B2*x + C2*t))*(-sech2)
                                                   + sq2*(A1*sin(B1*x) + A2*C2*cos(B2*x + C2*t)));
}
        
__device__ void fprime(const fp_t eta, fp_t* f)
{
    *f = 4. * eta * (eta - 1.) * (eta - 0.5);
}

__global__ void evolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new, fp_t* d_conc_lap,
                                 const fp_t dx, const fp_t dy, const fp_t dt,
                                 const fp_t elapsed,
                                 const int  nx, const int  ny, const int  nm,
                                 const fp_t A1, const fp_t A2, const fp_t B1, const fp_t B2,
                                 const fp_t C2, const fp_t kappa)
{
	int thr_x, thr_y, x, y;
    fp_t xx, yy;
    fp_t S, f;

	/* determine indices on which to operate */
	thr_x = threadIdx.x;
	thr_y = threadIdx.y;

	x = blockDim.x * blockIdx.x + thr_x;
	y = blockDim.y * blockIdx.y + thr_y;

	/* explicit Euler solution to the Allen-Cahn equation of motion */
	if (x < nx && y < ny) {
    	xx = dx * (x - nm/2);
    	yy = dy * (y - nm/2);
    	const fp_t eta = d_conc_old[nx * y + x];
        const fp_t lap = d_conc_lap[nx * y + x];
    	fprime(eta, &f);
    	source(xx, yy, elapsed, A1, A2, B1, B2, C2, kappa, &S);
		d_conc_new[nx * y + x] = eta
                               - dt * (f - kappa * lap)
                               + dt * S;
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void device_boundaries(fp_t* conc,
                       const int bx, const int by,
                       const int nx, const int ny, const int nm)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    conc, nx, ny, nm
	);
}

void device_convolution(fp_t* conc_old, fp_t* conc_lap,
                        const int bx, const int by,
                        const int nx, const int ny, const int nm)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    conc_old, conc_lap, nx, ny, nm
	);
}

void device_evolution(fp_t* conc_old, fp_t* conc_new, fp_t* conc_lap,
                      const int  bx, const int  by,
                      const fp_t dx, const fp_t dy, const fp_t dt,
                      const fp_t elapsed,
                      const int  nx, const int  ny, const int  nm,
                      const fp_t A1, const fp_t A2, 
                      const fp_t B1, const fp_t B2, 
                      const fp_t C2, const fp_t kappa)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	evolution_kernel<<<num_tiles,tile_size>>> (
	    conc_old, conc_new, conc_lap, dx, dy, dt, elapsed, nx, ny, nm, A1, A2, B1, B2, C2, kappa
	);
}

void read_out_result(fp_t** conc, fp_t* d_conc, const int nx, const int ny)
{
	hipMemcpy(conc[0], d_conc, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}

void cuda_evolution_solver(struct CudaData* dev, fp_t** conc_new,
                           const int  bx, const int  by,
                           const fp_t dx, const fp_t dy, const fp_t dt,
                           const fp_t elapsed, 
                           const int  nx, const int  ny, const int  nm,
						   const fp_t A1, const fp_t A2,
						   const fp_t B1, const fp_t B2,
						   const fp_t C2, const fp_t kappa,
                           struct Stopwatch* sw)
{
	double start_time;

	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	/* apply boundary conditions */
	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_old, nx, ny, nm
	);

	/* compute Laplacian */
	start_time = GetTimer();
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_old, dev->conc_lap, nx, ny, nm
	);
	sw->conv += GetTimer() - start_time;

	/* compute result */
	start_time = GetTimer();
	evolution_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_old, dev->conc_new, dev->conc_lap, dx, dy, dt, elapsed, nx, ny, nm, A1, A2, B1, B2, C2, kappa
	);
	sw->step += GetTimer() - start_time;
}
