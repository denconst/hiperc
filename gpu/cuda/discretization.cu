#include "hip/hip_runtime.h"
/*
	File: discretization.c
	Role: implementation of discretized mathematical operations with OpenMP threading and CUDA acceleration

	Questions/comments to trevor.keller@nist.gov
	Bugs/requests to https://github.com/usnistgov/phasefield-accelerator-benchmarks
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "diffusion.h"
}

/* CUDA allocates memory tiles on the GPU statically, so their sizes must be hard coded */
#define MAX_TILE_W 32
#define MAX_TILE_H 32
#define MAX_MASK_W 3
#define MAX_MASK_SIZE (MAX_MASK_W * MAX_MASK_W)

__constant__ double Mc[MAX_MASK_SIZE];

void set_threads(int n)
{
	omp_set_num_threads(n);
}

void five_point_Laplacian_stencil(double dx, double dy, double** M)
{
	M[0][1] =  1. / (dy * dy); /* up */
	M[1][0] =  1. / (dx * dx); /* left */
	M[1][1] = -2. * (dx*dx + dy*dy) / (dx*dx * dy*dy); /* middle */
	M[1][2] =  1. / (dx * dx); /* right */
	M[2][1] =  1. / (dy * dy); /* down */
}

void nine_point_Laplacian_stencil(double dx, double dy, double** M)
{
	M[0][0] =   1. / (6. * dx * dy);
	M[0][1] =   4. / (6. * dy * dy);
	M[0][2] =   1. / (6. * dx * dy);

	M[1][0] =   4. / (6. * dx * dx);
	M[1][1] = -10. * (dx*dx + dy*dy) / (6. * dx*dx * dy*dy);
	M[1][2] =   4. / (6. * dx * dx);

	M[2][0] =   1. / (6. * dx * dy);
	M[2][1] =   4. / (6. * dy * dy);
	M[2][2] =   1. / (6. * dx * dy);
}

void set_mask(double dx, double dy, int nm, double** M)
{
	five_point_Laplacian_stencil(dx, dy, M);
}

__global__ void convolution_kernel(double* A, double* C, int nx, int ny, int nm)
{
	/* Notes:
		* The source matrix (A) and destination matrix (C) must be identical in size
		* One CUDA core operates on one array index: there is no nested loop over matrix elements
		* The halo (nm/2 perimeter cells) in C are unallocated garbage
		* The same cells in A are boundary values, and contribute to the convolution
		* N_ds is the shared tile data array... dunno where the name comes from yet
	*/

	int i, j, tx, ty,
	    dst_row, dst_col, dst_tile_w, dst_tile_h,
	    src_row, src_col, src_tile_w, src_tile_h;
	double value=0.;

	/* source tile width includes the halo cells */
	src_tile_w = blockDim.x;
	src_tile_h = blockDim.y;

	/* destination tile width excludes the halo cells */
	dst_tile_w = src_tile_w - nm + 1;
	dst_tile_h = src_tile_h - nm + 1;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	dst_row = blockIdx.y * dst_tile_h + ty;
	dst_col = blockIdx.x * dst_tile_w + tx;

	src_row = dst_row - nm/2;
	src_col = dst_col - nm/2;

	/* copy tile from A: __shared__ gives access to all threads working on this tile */
	__shared__ double N_ds[MAX_TILE_H + MAX_MASK_W - 1][MAX_TILE_W + MAX_MASK_W - 1];

	if ((src_row >= 0) && (src_row < ny) &&
	    (src_col >= 0) && (src_col < nx)) {
		/* if src_row==0, then dst_row==nm/2: this is a halo row, still contributing to the output */
		N_ds[ty][tx] = A[src_row * nx + src_col];
	} else {
		/* points outside the halo should be switched off */
		N_ds[ty][tx] = 0.;
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (tx < dst_tile_w && ty < dst_tile_h) {
		for (j = 0; j < nm; j++) {
			for (i = 0; i < nm; i++) {
				value += Mc[j * nm + i] * N_ds[j+ty][i+tx];
			}
		}
		/* record value */
		if (dst_row < ny && dst_col < nx) {
			C[dst_row * nx + dst_col] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void compute_convolution(double** A, double** C, double** M, int nx, int ny, int nm, int bs)
{
	double* d_A, *d_C;

	if (bs > MAX_TILE_W) {
		printf("Error: requested block size %i exceeds the statically allocated array size.\n", bs);
		exit(-1);
	}

	/* allocate memory on device */
	hipMalloc((void **) &d_A, nx * ny * sizeof(double));
	hipMalloc((void **) &d_C, nx * ny * sizeof(double));

	/* transfer data from host in to device */
	hipMemcpy(d_A, A[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);

	/* transfer mask in to constant device memory */
	hipMemcpyToSymbol(HIP_SYMBOL(Mc), M[0], nm * nm * sizeof(double));

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs - nm/2, bs - nm/2, 1);
	dim3 blocks(ceil(double(nx)/threads.x)+1, ceil(double(ny)/threads.y)+1, 1);

	/* compute result */
	convolution_kernel<<<blocks, threads>>>(d_A, d_C, nx, ny, nm);

	/* transfer from device out from host */
	hipMemcpy(C[0], d_C, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_A);
	hipFree(d_C);
}

__global__ void diffusion_kernel(double* A, double* B, double* C,
                                 int nx, int ny, int nm, double D, double dt)
{
	int tx, ty, row, col;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	row = blockDim.y * blockIdx.y + ty;
	col = blockDim.x * blockIdx.x + tx;

	/* explicit Euler solution to the equation of motion */
	if (row < ny && col < nx) {
		B[row * nx + col] = A[row * nx + col] + dt * D * C[row * nx + col];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void solve_diffusion_equation(double** A, double** B, double** C,
                              int nx, int ny, int nm, int bs, double D, double dt, double* elapsed)
{
	double* d_A, *d_B, *d_C;

	/* allocate memory on device */
	hipMalloc((void **) &d_A, nx * ny * sizeof(double));
	hipMalloc((void **) &d_B, nx * ny * sizeof(double));
	hipMalloc((void **) &d_C, nx * ny * sizeof(double));

	/* transfer data from host in to device */
	hipMemcpy(d_A, A[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs - nm/2, bs - nm/2, 1);
	dim3 blocks(ceil(double(nx)/threads.x)+1, ceil(double(ny)/threads.y)+1, 1);

	/* compute result */
	diffusion_kernel<<<blocks, threads>>>(d_A, d_B, d_C, nx, ny, nm, D, dt);

	/* transfer from device out from host */
	hipMemcpy(B[0], d_B, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	*elapsed += dt;
}

void analytical_value(double x, double t, double D, double bc[2][2], double* c)
{
	*c = bc[1][0] * (1. - erf(x / sqrt(4. * D * t)));
}

void check_solution(double** A,
                    int nx, int ny, double dx, double dy, int nm, int bs,
                    double elapsed, double D, double bc[2][2], double* rss)
{
	double sum=0.;
	#pragma omp parallel reduction(+:sum)
	{
		int i, j;
		double r, cal, car, ca, cn, trss;

		#pragma omp for collapse(2)
		for (j = nm/2; j < ny-nm/2; j++) {
			for (i = nm/2; i < nx-nm/2; i++) {
				/* numerical solution */
				cn = A[j][i];

				/* shortest distance to left-wall source */
				r = (j < ny/2) ? dx * (i - nm/2) : sqrt(dx*dx * (i - nm/2) * (i - nm/2) + dy*dy * (j - ny/2) * (j - ny/2));
				analytical_value(r, elapsed, D, bc, &cal);

				/* shortest distance to right-wall source */
				r = (j >= ny/2) ? dx * (nx-nm/2-1 - i) : sqrt(dx*dx * (nx-nm/2-1 - i)*(nx-nm/2-1 - i) + dy*dy * (ny/2 - j)*(ny/2 - j));
				analytical_value(r, elapsed, D, bc, &car);

				/* superposition of analytical solutions */
				ca = cal + car;

				/* residual sum of squares (RSS) */
				trss = (ca - cn) * (ca - cn) / (double)((nx-nm/2-1) * (ny-nm/2-1));
				sum += trss;
			}
		}
	}

	*rss = sum;
}
