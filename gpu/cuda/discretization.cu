#include "hip/hip_runtime.h"
/*
	File: discretization.c
	Role: implementation of discretized mathematical operations with OpenMP threading and CUDA acceleration

	Questions/comments to trevor.keller@nist.gov
	Bugs/requests to https://github.com/usnistgov/phasefield-accelerator-benchmarks
*/

#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "diffusion.h"
}

/* CUDA allocates memory tiles on the GPU statically, so their sizes must be hard coded */
#define MAX_TILE_W 32
#define MAX_TILE_H 32
#define MAX_MASK_W 3
#define MAX_MASK_SIZE (MAX_MASK_W * MAX_MASK_W)

__constant__ double Mc[MAX_MASK_SIZE];

void set_threads(int n)
{
	omp_set_num_threads(n);
}

void five_point_Laplacian_stencil(double dx, double dy, double** M)
{
	M[0][0] =  0.;
	M[0][1] =  1. / (dy * dy); /* up */
	M[0][2] =  0.;

	M[1][0] =  1. / (dx * dx); /* left */
	M[1][1] = -2. * (dx*dx + dy*dy) / (dx*dx * dy*dy); /* middle */
	M[1][2] =  1. / (dx * dx); /* right */

	M[2][0] =  0.;
	M[2][1] =  1. / (dy * dy); /* down */
	M[2][2] =  0.;
}

void nine_point_Laplacian_stencil(double dx, double dy, double** M)
{
	M[0][0] =   1. / (6. * dx * dy);
	M[0][1] =   4. / (6. * dy * dy);
	M[0][2] =   1. / (6. * dx * dy);

	M[1][0] =   4. / (6. * dx * dx);
	M[1][1] = -10. * (dx*dx + dy*dy) / (6. * dx*dx * dy*dy);
	M[1][2] =   4. / (6. * dx * dx);

	M[2][0] =   1. / (6. * dx * dy);
	M[2][1] =   4. / (6. * dy * dy);
	M[2][2] =   1. / (6. * dx * dy);
}

void set_mask(double dx, double dy, int nm, double** M)
{
	five_point_Laplacian_stencil(dx, dy, M);
}

__global__ void convolution_kernel(double* A, double* C, int nx, int ny, int nm)
{
	/* Notes:
		* The source matrix (A) and destination matrix (C) must be identical in size
		* One CUDA core operates on one array index: there is no nested loop over matrix elements
		* The halo (nm/2 perimeter cells) in C are unallocated garbage
		* The same cells in A are boundary values, and contribute to the convolution
		* N_ds is the shared tile data array... dunno where the name comes from yet
	*/

	int i, j, tx, ty, dst_row, dst_col, src_row, src_col;
	double value=0.;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	dst_row = blockDim.y * blockIdx.y + ty;
	dst_col = blockDim.x * blockIdx.x + tx;

	src_row = dst_row - nm/2;
	src_col = dst_col - nm/2;

	/* copy tile from A: __shared__ gives access to all threads working on this tile */
	__shared__ double N_ds[MAX_TILE_H + MAX_MASK_W - 1][MAX_TILE_W + MAX_MASK_W - 1];

	__syncthreads();

	if ((dst_row > 0) && (src_row < ny) &&
	    (dst_col > 0) && (src_col < nx)) {
		/* if src_row==0, then dst_row==nm/2: this is a halo row, still contributing to the output */
		N_ds[ty][tx] = A[src_row*nx + src_col];
	} else {
		/* points outside the halo should be switched off */
		N_ds[ty][tx] = 0.;
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (tx < blockDim.x && ty < blockDim.y) {
		for (j = 0; j < nm; j++) {
			for (i = 0; i < nm; i++) {
				value += Mc[j*nm + i] * N_ds[j+ty][i+tx];
			}
		}

		/* record value */
		if ((dst_row > 0 && dst_row < ny-1) &&
		     dst_col > 0 && dst_col < nx-1) {
			C[dst_row*nx + dst_col] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void compute_convolution(double** A, double** C, double** M, int nx, int ny, int nm, int bs)
{
	double* d_A, *d_C;

	/* allocate memory on device */
	hipMalloc((void **) &d_A, nx * ny * sizeof(double));
	hipMalloc((void **) &d_C, nx * ny * sizeof(double));

	/* transfer data from host in to device */
	hipMemcpy(d_A, A[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);

	/* transfer mask in to constant device memory */
	hipMemcpyToSymbol(HIP_SYMBOL(Mc), M[0], nm * nm * sizeof(double));

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs, bs, 1);
	dim3 blocks(ceil(double(nx)/threads.x), ceil(double(ny)/threads.y), 1);

	/* compute result */
	convolution_kernel<<<blocks, threads>>>(d_A, d_C, nx, ny, nm);

	/* transfer from device out from host */
	hipMemcpy(C[0], d_C, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_A);
	hipFree(d_C);
}

__global__ void diffusion_kernel(double* A, double* B, double* C,
                                 int nx, int ny, int nm, int bs, double D, double dt)
{
	int tx, ty, row, col;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	row = blockDim.y * blockIdx.y + ty;
	col = blockDim.x * blockIdx.x + tx;

	/* explicit Euler solution to the equation of motion */
	if ((row > 0 && row < ny) &&
	    (col > 0 && col < nx)) {
		B[row*nx + col] = A[row*nx + col] + dt * D * C[row*nx + col];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void solve_diffusion_equation(double** A, double** B, double** C,
                              int nx, int ny, int nm, int bs, double D, double dt, double* elapsed)
{
	double* d_A, *d_B, *d_C;

	/* allocate memory on device */
	hipMalloc((void **) &d_A, nx * ny * sizeof(double));
	hipMalloc((void **) &d_B, nx * ny * sizeof(double));
	hipMalloc((void **) &d_C, nx * ny * sizeof(double));

	/* transfer data from host in to device */
	hipMemcpy(d_A, A[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C[0], nx * ny * sizeof(double), hipMemcpyHostToDevice);

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs, bs, 1);
	dim3 blocks(ceil(double(nx)/threads.x), ceil(double(ny)/threads.y), 1);

	/* compute result */
	diffusion_kernel<<<blocks, threads>>>(d_A, d_B, d_C, nx, ny, nm, bs, D, dt);

	/* transfer from device out from host */
	hipMemcpy(B[0], d_B, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	*elapsed += dt;
}

void check_solution(double** A,
                    int nx, int ny, double dx, double dy, int nm, int bs,
                    double elapsed, double D, double bc[2][2], double* rss)
{
	/* Not easily CUDA-able without a prefix-sum formulation */

	/* OpenCL does not have a GPU-based erf() definition, using Maclaurin series approximation */
	double sum=0.;
	#pragma omp parallel reduction(+:sum)
	{
		int i, j;
		double ca, cal, car, cn, poly_erf, r, trss, z, z2;

		#pragma omp for collapse(2)
		for (j = nm/2; j < ny-nm/2; j++) {
			for (i = nm/2; i < nx-nm/2; i++) {
				/* numerical solution */
				cn = A[j][i];

				/* shortest distance to left-wall source */
				r = (j < ny/2) ? dx * (i - nm/2) :
				    sqrt(dx*dx * (i - nm/2) * (i - nm/2) + dy*dy * (j - ny/2) * (j - ny/2));
				z = r / sqrt(4. * D * elapsed);
				z2 = z * z;
				poly_erf = (z > 1.5) ? 1. :
				           2. * z * (1. + z2 * (-1./3 + z2 * (1./10 + z2 * (-1./42 + z2 / 216)))) / sqrt(M_PI);
				cal = bc[1][0] * (1. - poly_erf);

				/* shortest distance to right-wall source */
				r = (j >= ny/2) ? dx * (nx-nm+1 - i) :
				    sqrt(dx*dx * (nx-nm+1 - i)*(nx-nm+1 - i) + dy*dy * (ny/2 - j)*(ny/2 - j));
				z = r / sqrt(4. * D * elapsed);
				z2 = z * z;
				poly_erf = (z > 1.5) ? 1. :
				           2. * z * (1. + z2 * (-1./3 + z2 * (1./10 + z2 * (-1./42 + z2 / 216)))) / sqrt(M_PI);
				car = bc[1][0] * (1. - poly_erf);

				/* superposition of analytical solutions */
				ca = cal + car;

				/* residual sum of squares (RSS) */
				trss = (ca - cn) * (ca - cn) / (double)((nx-nm+1) * (ny-nm+1));
				sum += trss;
			}
		}
	}

	*rss = sum;
}
