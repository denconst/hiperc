#include "hip/hip_runtime.h"
/*
	File: discretization.c
	Role: implementation of discretized mathematical operations with OpenMP threading and CUDA acceleration

	Questions/comments to trevor.keller@nist.gov
	Bugs/requests to https://github.com/usnistgov/phasefield-accelerator-benchmarks
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "diffusion.h"
}

/* CUDA allocates memory tiles on the GPU statically, so their sizes must be hard coded */
#define MAX_TILE_W 32
#define MAX_TILE_H 32
#define MAX_MASK_W 3
#define MAX_MASK_SIZE (MAX_MASK_W * MAX_MASK_W)

__constant__ fp_t Mc[MAX_MASK_SIZE];

void set_threads(int n)
{
	omp_set_num_threads(n);
}

void five_point_Laplacian_stencil(fp_t dx, fp_t dy, fp_t** mask_lap)
{
	mask_lap[0][1] =  1. / (dy * dy); /* up */
	mask_lap[1][0] =  1. / (dx * dx); /* left */
	mask_lap[1][1] = -2. * (dx*dx + dy*dy) / (dx*dx * dy*dy); /* middle */
	mask_lap[1][2] =  1. / (dx * dx); /* right */
	mask_lap[2][1] =  1. / (dy * dy); /* down */
}

void nine_point_Laplacian_stencil(fp_t dx, fp_t dy, fp_t** mask_lap)
{
	mask_lap[0][0] =   1. / (6. * dx * dy);
	mask_lap[0][1] =   4. / (6. * dy * dy);
	mask_lap[0][2] =   1. / (6. * dx * dy);

	mask_lap[1][0] =   4. / (6. * dx * dx);
	mask_lap[1][1] = -10. * (dx*dx + dy*dy) / (6. * dx*dx * dy*dy);
	mask_lap[1][2] =   4. / (6. * dx * dx);

	mask_lap[2][0] =   1. / (6. * dx * dy);
	mask_lap[2][1] =   4. / (6. * dy * dy);
	mask_lap[2][2] =   1. / (6. * dx * dy);
}

void slow_nine_point_Laplacian_stencil(fp_t dx, fp_t dy, fp_t** mask_lap)
{
	/* 4x4 mask, 9 values, truncation error O(dx^4)
	   Provided for testing and demonstration of scalability, only:
	   as the name indicates, this 9-point stencil is computationally
	   more expensive than the 3x3 version. If your code requires O(dx^4)
	   accuracy, please use nine_point_Laplacian_stencil. */

	mask_lap[0][2] = -1. / (12. * dy * dy);

	mask_lap[1][2] =  4. / (3. * dy * dy);

	mask_lap[2][0] = -1. / (12. * dx * dx);
	mask_lap[2][1] =  4. / (3. * dx * dx);
	mask_lap[2][2] = -5. * (dx*dx + dy*dy) / (2. * dx*dx * dy*dy);
	mask_lap[2][3] =  4. / (3. * dx * dx);
	mask_lap[2][4] = -1. / (12. * dx * dx);

	mask_lap[3][2] =  4. / (3. * dy * dy);

	mask_lap[4][2] = -1. / (12. * dy * dy);
}

void set_mask(fp_t dx, fp_t dy, int nm, fp_t** mask_lap)
{
	five_point_Laplacian_stencil(dx, dy, mask_lap);
}

__global__ void convolution_kernel(fp_t* conc_old, fp_t* conc_lap, int nx, int ny, int nm)
{
	/* Notes:
		* The source matrix (conc_old) and destination matrix (conc_lap) must be identical in size
		* One CUDA core operates on one array index: there is no nested loop over matrix elements
		* The halo (nm/2 perimeter cells) in conc_lap are unallocated garbage
		* The same cells in conc_old are boundary values, and contribute to the convolution
		* N_ds is the shared tile data array... dunno where the name comes from yet
	*/

	int i, j, tx, ty,
	    dst_row, dst_col, dst_tile_w, dst_tile_h,
	    src_row, src_col, src_tile_w, src_tile_h;
	fp_t value=0.;

	/* source tile width includes the halo cells */
	src_tile_w = blockDim.x;
	src_tile_h = blockDim.y;

	/* destination tile width excludes the halo cells */
	dst_tile_w = src_tile_w - nm + 1;
	dst_tile_h = src_tile_h - nm + 1;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	dst_row = blockIdx.y * dst_tile_h + ty;
	dst_col = blockIdx.x * dst_tile_w + tx;

	src_row = dst_row - nm/2;
	src_col = dst_col - nm/2;

	/* copy tile from conc_old: __shared__ gives access to all threads working on this tile */
	__shared__ fp_t N_ds[MAX_TILE_H + MAX_MASK_W - 1][MAX_TILE_W + MAX_MASK_W - 1];

	if ((src_row >= 0) && (src_row < ny) &&
	    (src_col >= 0) && (src_col < nx)) {
		/* if src_row==0, then dst_row==nm/2: this is a halo row, still contributing to the output */
		N_ds[ty][tx] = conc_old[src_row * nx + src_col];
	} else {
		/* points outside the halo should be switched off */
		N_ds[ty][tx] = 0.;
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (tx < dst_tile_w && ty < dst_tile_h) {
		for (j = 0; j < nm; j++) {
			for (i = 0; i < nm; i++) {
				value += Mc[j * nm + i] * N_ds[j+ty][i+tx];
			}
		}
		/* record value */
		if (dst_row < ny && dst_col < nx) {
			conc_lap[dst_row * nx + dst_col] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void compute_convolution(fp_t** conc_old, fp_t** conc_lap, fp_t** mask_lap, int nx, int ny, int nm, int bs)
{
	fp_t* d_conc_old, *d_conc_lap;

	if (bs > MAX_TILE_W) {
		printf("Error: requested block size %i exceeds the statically allocated array size.\n", bs);
		exit(-1);
	}

	/* allocate memory on device */
	hipMalloc((void **) &d_conc_old, nx * ny * sizeof(fp_t));
	hipMalloc((void **) &d_conc_lap, nx * ny * sizeof(fp_t));

	/* transfer data from host in to device */
	hipMemcpy(d_conc_old, conc_old[0], nx * ny * sizeof(fp_t), hipMemcpyHostToDevice);

	/* transfer mask in to constant device memory */
	hipMemcpyToSymbol(HIP_SYMBOL(Mc), mask_lap[0], nm * nm * sizeof(fp_t));

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs - nm/2, bs - nm/2, 1);
	dim3 blocks(ceil(fp_t(nx)/threads.x)+1, ceil(fp_t(ny)/threads.y)+1, 1);

	/* compute result */
	convolution_kernel<<<blocks, threads>>>(d_conc_old, d_conc_lap, nx, ny, nm);

	/* transfer from device out from host */
	hipMemcpy(conc_lap[0], d_conc_lap, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_conc_old);
	hipFree(d_conc_lap);
}

__global__ void diffusion_kernel(fp_t* conc_old, fp_t* conc_new, fp_t* conc_lap,
                                 int nx, int ny, int nm, fp_t D, fp_t dt)
{
	int tx, ty, row, col;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	row = blockDim.y * blockIdx.y + ty;
	col = blockDim.x * blockIdx.x + tx;

	/* explicit Euler solution to the equation of motion */
	if (row < ny && col < nx) {
		conc_new[row * nx + col] = conc_old[row * nx + col] + dt * D * conc_lap[row * nx + col];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void solve_diffusion_equation(fp_t** conc_old, fp_t** conc_new, fp_t** conc_lap,
                              int nx, int ny, int nm, int bs, fp_t D, fp_t dt, fp_t* elapsed)
{
	fp_t* d_conc_old, *d_conc_new, *d_conc_lap;

	/* allocate memory on device */
	hipMalloc((void **) &d_conc_old, nx * ny * sizeof(fp_t));
	hipMalloc((void **) &d_conc_new, nx * ny * sizeof(fp_t));
	hipMalloc((void **) &d_conc_lap, nx * ny * sizeof(fp_t));

	/* transfer data from host in to device */
	hipMemcpy(d_conc_old, conc_old[0], nx * ny * sizeof(fp_t), hipMemcpyHostToDevice);
	hipMemcpy(d_conc_lap, conc_lap[0], nx * ny * sizeof(fp_t), hipMemcpyHostToDevice);

	/* divide matrices into blocks of (bs x bs) threads */
	dim3 threads(bs - nm/2, bs - nm/2, 1);
	dim3 blocks(ceil(fp_t(nx)/threads.x)+1, ceil(fp_t(ny)/threads.y)+1, 1);

	/* compute result */
	diffusion_kernel<<<blocks, threads>>>(d_conc_old, d_conc_new, d_conc_lap, nx, ny, nm, D, dt);

	/* transfer from device out from host */
	hipMemcpy(conc_new[0], d_conc_new, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_conc_old);
	hipFree(d_conc_new);
	hipFree(d_conc_lap);

	*elapsed += dt;
}

void analytical_value(fp_t x, fp_t t, fp_t D, fp_t bc[2][2], fp_t* c)
{
	*c = bc[1][0] * (1. - erf(x / sqrt(4. * D * t)));
}

void check_solution(fp_t** conc_new,
                    int nx, int ny, fp_t dx, fp_t dy, int nm, int bs,
                    fp_t elapsed, fp_t D, fp_t bc[2][2], fp_t* rss)
{
	fp_t sum=0.;
	#pragma omp parallel reduction(+:sum)
	{
		int i, j;
		fp_t r, cal, car, ca, cn, trss;

		#pragma omp for collapse(2)
		for (j = nm/2; j < ny-nm/2; j++) {
			for (i = nm/2; i < nx-nm/2; i++) {
				/* numerical solution */
				cn = conc_new[j][i];

				/* shortest distance to left-wall source */
				r = (j < ny/2) ? dx * (i - nm/2) : sqrt(dx*dx * (i - nm/2) * (i - nm/2) + dy*dy * (j - ny/2) * (j - ny/2));
				analytical_value(r, elapsed, D, bc, &cal);

				/* shortest distance to right-wall source */
				r = (j >= ny/2) ? dx * (nx-1-nm/2 - i) : sqrt(dx*dx * (nx-1-nm/2 - i)*(nx-1-nm/2 - i) + dy*dy * (ny/2 - j)*(ny/2 - j));
				analytical_value(r, elapsed, D, bc, &car);

				/* superposition of analytical solutions */
				ca = cal + car;

				/* residual sum of squares (RSS) */
				trss = (ca - cn) * (ca - cn) / (fp_t)((nx-1-nm/2) * (ny-1-nm/2));
				sum += trss;
			}
		}
	}

	*rss = sum;
}
