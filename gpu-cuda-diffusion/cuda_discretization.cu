#include "hip/hip_runtime.h"
/**********************************************************************************
 This file is part of Phase-field Accelerator Benchmarks, written by Trevor Keller
 and available from https://github.com/usnistgov/phasefield-accelerator-benchmarks.

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "boundaries.h"
#include "discretization.h"
#include "numerics.h"
#include "timer.h"
}

/**
 \brief Maximum width of an input tile, including halo cells, for GPU memory allocation
*/
#define MAX_TILE_W 32

/**
 \brief Maximum height of an input tile, including halo cells, for GPU memory allocation
*/
#define MAX_TILE_H 32

/**
 \brief Convolution mask array on the GPU, allocated in protected memory
 \fn fp_t Mc[MAX_MASK_W * MAX_MASK_H]
*/
__constant__ fp_t Mc[MAX_MASK_W * MAX_MASK_H];

/**
 \brief Tiled convolution algorithm for execution on the GPU
 \fn void convolution_kernel(fp_t* conc_old, fp_t* conc_lap, int nx, int ny, int nm)

 This function accesses 1D data rather than the 2D array representation of the
 scalar composition field, mapping into 2D tiles on the GPU with halo cells
 before computing the convolution. Note:
 - The source matrix (\a conc_old) and destination matrix (\a conc_lap) must be identical in size
 - One CUDA core operates on one array index: there is no nested loop over matrix elements
 - The halo (\a nm/2 perimeter cells) in \a conc_lap are unallocated garbage
 - The same cells in \a conc_old are boundary values, and contribute to the convolution
 - \a conc_tile is the shared tile of input data, accessible by all threads in this block
*/
__global__ void convolution_kernel(fp_t* conc_old, fp_t* conc_lap, int nx, int ny, int nm)
{
	int i, j, tx, ty,
	    dst_row, dst_col, dst_tile_w, dst_tile_h,
	    src_row, src_col, src_tile_w, src_tile_h;
	fp_t value=0.;

	/* source tile width includes the halo cells */
	src_tile_w = blockDim.x;
	src_tile_h = blockDim.y;

	/* destination tile width excludes the halo cells */
	dst_tile_w = src_tile_w - nm + 1;
	dst_tile_h = src_tile_h - nm + 1;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	dst_row = blockIdx.y * dst_tile_h + ty;
	dst_col = blockIdx.x * dst_tile_w + tx;

	src_row = dst_row - nm/2;
	src_col = dst_col - nm/2;

	/* copy tile from conc_old: __shared__ gives access to all threads working on this tile	*/
	__shared__ fp_t conc_tile[MAX_TILE_H + MAX_MASK_H - 1][MAX_TILE_W + MAX_MASK_W - 1];

	if ((src_row >= 0) && (src_row < ny) &&
	    (src_col >= 0) && (src_col < nx)) {
		/* if src_row==0, then dst_row==nm/2: this is a halo row, still contributing to the output */
		conc_tile[ty][tx] = conc_old[src_row * nx + src_col];
	} else {
		/* points outside the halo should be switched off */
		conc_tile[ty][tx] = 0.;
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (tx < dst_tile_w && ty < dst_tile_h) {
		for (j = 0; j < nm; j++) {
			for (i = 0; i < nm; i++) {
				value += Mc[j * nm + i] * conc_tile[j+ty][i+tx];
			}
		}
		/* record value */
		if (dst_row < ny && dst_col < nx) {
			conc_lap[dst_row * nx + dst_col] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void compute_convolution(fp_t** conc_old, fp_t** conc_lap, fp_t** mask_lap,
                         int nx, int ny, int nm)
{
}

/**
 \brief Vector addition algorithm for execution on the GPU
 \fn void diffusion_kernel(fp_t* conc_old, fp_t* conc_new, fp_t* conc_lap, int nx, int ny, int nm, fp_t D, fp_t dt)

 This function accesses 1D data rather than the 2D array representation of the
 scalar composition field
*/
__global__ void diffusion_kernel(fp_t* conc_old, fp_t* conc_new, fp_t* conc_lap,
                                 int nx, int ny, int nm, fp_t D, fp_t dt)
{
	int tx, ty, row, col;

	/* determine indices on which to operate */
	tx = threadIdx.x;
	ty = threadIdx.y;

	row = blockDim.y * blockIdx.y + ty;
	col = blockDim.x * blockIdx.x + tx;

	/* explicit Euler solution to the equation of motion */
	if (row < ny && col < nx) {
		conc_new[row * nx + col] = conc_old[row * nx + col] + dt * D * conc_lap[row * nx + col];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void solve_diffusion_equation(fp_t** conc_old, fp_t** conc_new, fp_t** conc_lap,
                              fp_t** mask_lap, int nx, int ny, int nm,
                              fp_t bc[2][2], fp_t D, fp_t dt, fp_t* elapsed,
                              struct Stopwatch* sw)
{
	fp_t* d_conc_old, *d_conc_new, *d_conc_lap;
	double start_time;

	apply_boundary_conditions(conc_old, nx, ny, nm, bc);

	start_time = GetTimer();

	/* allocate memory on device */
	hipMalloc((void **) &d_conc_old, nx * ny * sizeof(fp_t));
	hipMalloc((void **) &d_conc_lap, nx * ny * sizeof(fp_t));

	/* divide matrices into blocks of (MAX_TILE_W x MAX_TILE_W) threads */
	dim3 threads(MAX_TILE_W - nm/2, MAX_TILE_W - nm/2, 1);
	dim3 blocks(ceil(fp_t(nx)/threads.x)+1, ceil(fp_t(ny)/threads.y)+1, 1);

	/* transfer mask in to constant device memory */
	hipMemcpyToSymbol(HIP_SYMBOL(Mc), mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer data from host in to device */
	hipMemcpy(d_conc_old, conc_old[0], nx * ny * sizeof(fp_t), hipMemcpyHostToDevice);

	/* compute Laplacian */
	convolution_kernel<<<blocks, threads>>>(d_conc_old, d_conc_lap, nx, ny, nm);
	sw->conv += GetTimer() - start_time;

	/* compute result */
	start_time = GetTimer();
	hipMalloc((void **) &d_conc_new, nx * ny * sizeof(fp_t));
	diffusion_kernel<<<blocks, threads>>>(d_conc_old, d_conc_new, d_conc_lap, nx, ny, nm, D, dt);

	/* transfer from device out to host */
	hipMemcpy(conc_new[0], d_conc_new, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	sw->step += GetTimer() - start_time;

	/* free memory on device */
	hipFree(d_conc_old);
	hipFree(d_conc_new);
	hipFree(d_conc_lap);

	*elapsed += dt;
}

void check_solution(fp_t** conc_new, fp_t** conc_lap, int nx, int ny,
                    fp_t dx, fp_t dy, int nm, fp_t elapsed, fp_t D,
                    fp_t bc[2][2], fp_t* rss)
{
	fp_t sum=0.;

	#pragma omp parallel reduction(+:sum)
	{
		int i, j;
		fp_t r, cal, car, ca, cn;

		#pragma omp for collapse(2) private(ca,cal,car,cn,i,j,r)
		for (j = nm/2; j < ny-nm/2; j++) {
			for (i = nm/2; i < nx-nm/2; i++) {
				/* numerical solution */
				cn = conc_new[j][i];

				/* shortest distance to left-wall source */
				r = distance_point_to_segment(dx * (nm/2), dy * (nm/2),
				                              dx * (nm/2), dy * (ny/2),
				                              dx * i, dy * j);
				analytical_value(r, elapsed, D, bc, &cal);

				/* shortest distance to right-wall source */
				r = distance_point_to_segment(dx * (nx-1-nm/2), dy * (ny/2),
				                              dx * (nx-1-nm/2), dy * (ny-1-nm/2),
				                              dx * i, dy * j);
				analytical_value(r, elapsed, D, bc, &car);

				/* superposition of analytical solutions */
				ca = cal + car;

				/* residual sum of squares (RSS) */
				conc_lap[j][i] = (ca - cn) * (ca - cn) / (fp_t)((nx-1-nm/2) * (ny-1-nm/2));
			}
		}

		#pragma omp for collapse(2) private(i,j)
		for (j = nm/2; j < ny-nm/2; j++) {
			for (i = nm/2; i < nx-nm/2; i++) {
				sum += conc_lap[j][i];
			}
		}
	}

	*rss = sum;
}
