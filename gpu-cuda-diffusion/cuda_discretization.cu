#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "cuda_data.h"
#include "numerics.h"
}

#include "cuda_kernels.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__global__ void convolution_kernel(fp_t* d_conc_old,
                                   fp_t* d_conc_lap,
                                   const int nx,
                                   const int ny,
                                   const int nm)
{
	int dst_x, dst_y, dst_nx, dst_ny;
	int src_x, src_y, src_nx, src_ny;
	int til_x, til_y, til_nx;
	fp_t value=0.;

	/* source and tile width include the halo cells */
	src_nx = blockDim.x;
	src_ny = blockDim.y;
	til_nx = src_nx;

	/* destination width excludes the halo cells */
	dst_nx = src_nx - nm + 1;
	dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	til_x = threadIdx.x;
	til_y = threadIdx.y;

	dst_x = blockIdx.x * dst_nx + til_x;
	dst_y = blockIdx.y * dst_ny + til_y;

	src_x = dst_x - nm/2;
	src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		if (dst_y < ny && dst_x < nx) {
			d_conc_lap[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__global__ void diffusion_kernel(fp_t* d_conc_old,
                                 fp_t* d_conc_new,
                                 fp_t* d_conc_lap,
                                 const int nx,
                                 const int ny,
                                 const int nm,
                                 const fp_t D,
                                 const fp_t dt)
{
	int thr_x, thr_y, x, y;

	/* determine indices on which to operate */
	thr_x = threadIdx.x;
	thr_y = threadIdx.y;

	x = blockDim.x * blockIdx.x + thr_x;
	y = blockDim.y * blockIdx.y + thr_y;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		d_conc_new[nx * y + x] = d_conc_old[nx * y + x]
		              + dt * D * d_conc_lap[nx * y + x];
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void compute_convolution_tiled(fp_t* d_conc_old, fp_t* d_conc_lap,
                         const int nx, const int ny, const int nm,
                         const int bx, const int by)
{
  dim3 tile_size(bx, by, 1);
  dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
                 ceil(float(ny) / (tile_size.y - nm + 1)),
                 1);
  size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

  convolution_kernel<<<num_tiles,tile_size,buf_size>>> (d_conc_old,
                                                        d_conc_lap,
                                                        nx, ny, nm);

}

void update_composition_tiled(fp_t* d_conc_old, fp_t* d_conc_lap, fp_t* d_conc_new,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by, const fp_t D, const fp_t dt)
{
  dim3 tile_size(bx, by, 1);
  dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
                 ceil(float(ny) / (tile_size.y - nm + 1)),
                 1);

  diffusion_kernel<<<num_tiles,tile_size>>> (d_conc_old,
                                             d_conc_new,
                                             d_conc_lap,
                                             nx, ny, nm, D, dt);	
}

void read_out_result(fp_t** conc, fp_t* d_conc, const int nx, const int ny)
{
  hipMemcpy(conc[0], d_conc, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
}

/**
 \brief Reference showing how to invoke the convolution kernel.

 A stand-alone function like this incurs the cost of host-to-device data
 transfer each time it is called: it is a teaching tool, not reusable code.
 It is the basis for cuda_diffusion_solver(), which achieves much better
 performance by bundling CUDA kernels together and intelligently managing
 data transfers between the host (CPU) and device (GPU).
*/
void standalone_convolution(fp_t** conc_old, fp_t** conc_lap, fp_t** mask_lap,
                         const int bx, const int by,
                         const int nm,
                         const int nx, const int ny)
{
	fp_t* d_conc_old, *d_conc_lap;

	/* allocate memory on device */
	hipMalloc((void **) &d_conc_old, nx * ny * sizeof(fp_t));
	hipMalloc((void **) &d_conc_lap, nx * ny * sizeof(fp_t));

	/* divide matrices into blocks of TILE_W * TILE_H threads */
	dim3 tile_size(bx,
	               by,
	               1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.x + nm) * sizeof(fp_t);

	/* transfer mask in to constant device memory */
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer data from host in to device */
	hipMemcpy(d_conc_old, conc_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);

	/* compute Laplacian */
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
		d_conc_old, d_conc_lap, nx, ny, nm
	);

	/* transfer from device out to host */
	hipMemcpy(conc_lap[0], d_conc_lap, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(d_conc_old);
	hipFree(d_conc_lap);
}
